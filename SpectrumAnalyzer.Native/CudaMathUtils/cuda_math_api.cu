#include "hip/hip_runtime.h"
// api.cpp (host code)
#include "cuda_math.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>

//chat gpt generated shit... need to test...

__global__ void saxpy_kernel(float a, const float* x, const float* y, float* out, int n);

__global__ void k_fftshift_interleaved(float* x, int n);
__global__ void k_power(const float* __restrict__ x, float* __restrict__ p, int n);
__global__ void k_power_db(const float* __restrict__ x, float* __restrict__ p_db, int n, float floor_db);
__global__ void k_fftshift_interleaved(float* x, int n);

static inline int checkCuda(hipError_t st, const char* where){
    if (st != hipSuccess){ std::fprintf(stderr,"[CUDA]%s: %s\n",where,hipGetErrorString(st)); return 1; }
    return 0;
}
static inline int checkCufft(hipfftResult st, const char* where){
    if (st != HIPFFT_SUCCESS){ std::fprintf(stderr,"[cuFFT]%s: %d\n",where,st); return 2; }
    return 0;
}

void saxpy(float a, const float* x_h, const float* y_h, float* out_h, int n) {
    float *x_d, *y_d, *out_d;
    size_t bytes = n * sizeof(float);
    hipMalloc(&x_d, bytes); hipMalloc(&y_d, bytes); hipMalloc(&out_d, bytes);
    hipMemcpy(x_d, x_h, bytes, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, bytes, hipMemcpyHostToDevice);

    dim3 blk(256), grd((n + blk.x - 1)/blk.x);
    saxpy_kernel<<<grd, blk>>>(a, x_d, y_d, out_d, n);
    hipDeviceSynchronize();

    hipMemcpy(out_h, out_d, bytes, hipMemcpyDeviceToHost);
    hipFree(x_d); hipFree(y_d); hipFree(out_d);
}

int iq_fft_c2c_forward(const float* in_host, float* out_host, int n){
    if (!in_host || !out_host || n<=0) return 3;
    size_t bytes = sizeof(float)*2*n;
    hipfftComplex *d_in=nullptr, *d_out=nullptr;
    if (int e=checkCuda(hipMalloc(&d_in, bytes), "hipMalloc d_in")) return e;
    if (int e=checkCuda(hipMalloc(&d_out,bytes), "hipMalloc d_out")) return e;
    if (int e=checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    hipfftHandle plan;
    if (int e=checkCufft(hipfftPlan1d(&plan, n, HIPFFT_C2C, 1), "plan C2C")) return e;
    if (int e=checkCufft(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD), "exec C2C FWD")) return e;
    if (int e=checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e=checkCuda(hipMemcpy(out_host, d_out, bytes, hipMemcpyDeviceToHost), "D2H")) return e;
    hipfftDestroy(plan);
    hipFree(d_in); hipFree(d_out);
    return 0;
}

int iq_fft_c2c_forward2(const float* in_host, float* out_host, int n) {
    if (!in_host || !out_host || n <= 0) return 3;

    // Interleaved complex: hipfftComplex == float2
    size_t bytes = sizeof(float) * 2 * n;
    hipfftComplex* d_in  = nullptr;
    hipfftComplex* d_out = nullptr;

    if (int e = checkCuda(hipMalloc(&d_in,  bytes), "hipMalloc d_in")) return e;
    if (int e = checkCuda(hipMalloc(&d_out, bytes), "hipMalloc d_out")) return e;
    if (int e = checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D in")) return e;

    hipfftHandle plan;
    if (int e = checkCufft(hipfftPlan1d(&plan, n, HIPFFT_C2C, 1), "hipfftPlan1d C2C")) return e;

    if (int e = checkCufft(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD), "hipfftExecC2C FWD")) return e;
    if (int e = checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize")) return e;

    if (int e = checkCuda(hipMemcpy(out_host, d_out, bytes, hipMemcpyDeviceToHost), "D2H out")) return e;

    hipfftDestroy(plan);
    hipFree(d_in); hipFree(d_out);
    return 0;
}

int iq_power_spectrum(const float* in_host, float* out_host, int n){
    if (!in_host || !out_host || n<=0) return 3;
    size_t bytes = sizeof(float)*2*n;
    float *d_in=nullptr; float *d_pow=nullptr;
    if (int e=checkCuda(hipMalloc(&d_in, bytes), "malloc d_in")) return e;
    if (int e=checkCuda(hipMalloc(&d_pow, sizeof(float)*n), "malloc d_pow")) return e;
    if (int e=checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    dim3 blk(256), grd((n+255)/256);
    k_power<<<grd,blk>>>(d_in, d_pow, n);
    if (int e=checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e=checkCuda(hipMemcpy(out_host, d_pow, sizeof(float)*n, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d_in); hipFree(d_pow);
    return 0;
}

int iq_power_db(const float* in_host, float* out_host, int n, float floor_db){
    if (!in_host || !out_host || n<=0) return 3;
    size_t bytes = sizeof(float)*2*n;
    float *d_in=nullptr; float *d_db=nullptr;
    if (int e=checkCuda(hipMalloc(&d_in, bytes), "malloc d_in")) return e;
    if (int e=checkCuda(hipMalloc(&d_db, sizeof(float)*n), "malloc d_db")) return e;
    if (int e=checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    dim3 blk(256), grd((n+255)/256);
    k_power_db<<<grd,blk>>>(d_in, d_db, n, floor_db);
    if (int e=checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e=checkCuda(hipMemcpy(out_host, d_db, sizeof(float)*n, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d_in); hipFree(d_db);
    return 0;
}

int iq_fftshift_inplace(float* io_host, int n){
    if (!io_host || n<=0) return 3;
    size_t bytes = sizeof(float)*2*n;
    float *d=nullptr;
    if (int e=checkCuda(hipMalloc(&d, bytes), "malloc d")) return e;
    if (int e=checkCuda(hipMemcpy(d, io_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    dim3 blk(256), grd(((n/2)+255)/256);
    k_fftshift_interleaved<<<grd,blk>>>(d, n);
    if (int e=checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e=checkCuda(hipMemcpy(io_host, d, bytes, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d);
    return 0;
}
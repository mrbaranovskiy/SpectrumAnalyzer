#include "hip/hip_runtime.h"
// api.cpp (host code)
#include "cuda_math.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>

//chat gpt generated shit... need to test...

__global__ void saxpy_kernel(float a, const float* x, const float* y, float* out, int n);

__global__ void k_fftshift_interleaved(float* x, int n);
__global__ void k_power(const float* __restrict__ x, float* __restrict__ p, int n);
__global__ void k_power_db(const float* __restrict__ x, float* __restrict__ p_db, int n, float floor_db);
__global__ void k_fftshift_interleaved(float* x, int n);
__global__ void k_scale(float* __restrict__ freqs, int N, float Fs);
__global__ void k_power_db_real(const float* __restrict__ x,
                                float* __restrict__ p_db,
                                int n,
                                float floor_db);

static inline int checkCuda(hipError_t st, const char* where)
{
    if (st != hipSuccess)
    {
        std::fprintf(stderr, "[CUDA]%s: %s\n", where, hipGetErrorString(st));
        return 1;
    }
    return 0;
}

static inline int checkCufft(hipfftResult st, const char* where)
{
    if (st != HIPFFT_SUCCESS)
    {
        std::fprintf(stderr, "[cuFFT]%s: %d\n", where, st);
        return 2;
    }
    return 0;
}

void saxpy(float a, const float* x_h, const float* y_h, float* out_h, int n)
{
    float *x_d, *y_d, *out_d;
    size_t bytes = n * sizeof(float);
    hipMalloc(&x_d, bytes);
    hipMalloc(&y_d, bytes);
    hipMalloc(&out_d, bytes);
    hipMemcpy(x_d, x_h, bytes, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, bytes, hipMemcpyHostToDevice);

    dim3 blk(256), grd((n + blk.x - 1) / blk.x);
    saxpy_kernel<<<grd, blk>>>(a, x_d, y_d, out_d, n);
    hipDeviceSynchronize();

    hipMemcpy(out_h, out_d, bytes, hipMemcpyDeviceToHost);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(out_d);
}

int iq_fft_c2c_forward(const float* in_host, float* out_host, int n)
{
    if (!in_host || !out_host || n <= 0) return 3;
    size_t bytes = sizeof(float) * 2 * n;
    hipfftComplex *d_in = nullptr, *d_out = nullptr;
    if (int e = checkCuda(hipMalloc(&d_in, bytes), "hipMalloc d_in")) return e;
    if (int e = checkCuda(hipMalloc(&d_out, bytes), "hipMalloc d_out")) return e;
    if (int e = checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    hipfftHandle plan;
    if (int e = checkCufft(hipfftPlan1d(&plan, n, HIPFFT_C2C, 1), "plan C2C")) return e;
    if (int e = checkCufft(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD), "exec C2C FWD")) return e;
    if (int e = checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e = checkCuda(hipMemcpy(out_host, d_out, bytes, hipMemcpyDeviceToHost), "D2H")) return e;
    hipfftDestroy(plan);
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}

int iq_fft_c2r_forward(const float* in_host, float* out_host, int n)
{
    if (!in_host || !out_host || n <= 0) return 3;

    const int n_complex = n / 2 + 1; // Hermitian packed length
    const size_t bytes_in = sizeof(float) * 2 * n_complex; // interleaved complex
    const size_t bytes_out = sizeof(float) * n; // real

    hipfftComplex* d_in = nullptr;
    float* d_out = nullptr;

    if (int e = checkCuda(hipMalloc(&d_in, bytes_in), "hipMalloc d_in")) return e;
    if (int e = checkCuda(hipMalloc(&d_out, bytes_out), "hipMalloc d_out")) return e;
    if (int e = checkCuda(hipMemcpy(d_in, in_host, bytes_in, hipMemcpyHostToDevice), "H2D in")) return e;

    hipfftHandle plan;
    if (int e = checkCufft(hipfftPlan1d(&plan, n, HIPFFT_C2R, 1), "hipfftPlan1d C2R")) return e;

    if (int e = checkCufft(hipfftExecC2R(plan, d_in, d_out), "hipfftExecC2R")) return e;
    if (int e = checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize")) return e;

    // NOTE: cuFFT C2R is unnormalized. If you want unitary IFFT, divide by n.
    if (int e = checkCuda(hipMemcpy(out_host, d_out, bytes_out, hipMemcpyDeviceToHost), "D2H out")) return e;

    hipfftDestroy(plan);
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}

int iq_power_spectrum(const float* in_host, float* out_host, int n)
{
    if (!in_host || !out_host || n <= 0) return 3;
    size_t bytes = sizeof(float) * 2 * n;
    float* d_in = nullptr;
    float* d_pow = nullptr;

    if (int e = checkCuda(hipMalloc(&d_in, bytes), "malloc d_in")) return e;
    if (int e = checkCuda(hipMalloc(&d_pow, sizeof(float) * n), "malloc d_pow")) return e;
    if (int e = checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;


    dim3 blk(256), grd((n + 255) / 256);
    k_power<<<grd,blk>>>(d_in, d_pow, n);
    if (int e = checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e = checkCuda(hipMemcpy(out_host, d_pow, sizeof(float) * n, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d_in);
    hipFree(d_pow);
    return 0;
}

int k_scale_r(float* out_host, float N, float Fs)
{
    float* d_freqs;
    hipMalloc(&d_freqs, N * sizeof(float));

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    k_scale<<<grid, block>>>(d_freqs, N, Fs);

    if (int e = checkCuda(hipDeviceSynchronize(), "sync")) return e;
    if (int e = checkCuda(hipMemcpy(out_host, d_freqs, sizeof(float) * N, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d_freqs);
}


// power db for complex numbers
int iq_power_db(const float* in_host, float* out_host, int n, float floor_db)
{
    if (!in_host || !out_host || n <= 0) return 3;
    size_t bytes = sizeof(float) * 2 * n;
    float* d_in = nullptr;
    float* d_db = nullptr;
    if (int e = checkCuda(hipMalloc(&d_in, bytes), "malloc d_in")) return e;
    if (int e = checkCuda(hipMalloc(&d_db, sizeof(float) * n), "malloc d_db")) return e;
    if (int e = checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    dim3 blk(256), grd((n + 255) / 256);
    k_power_db<<<grd,blk>>>(d_in, d_db, n, floor_db);
    if (int e = checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e = checkCuda(hipMemcpy(out_host, d_db, sizeof(float) * n, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d_in);
    hipFree(d_db);
    return 0;
}

//power db for real numbers
int iq_power_db_real(const float* in_host, float* out_host, int n, float floor_db)
{
    if (!in_host || !out_host || n <= 0)
        return 3;

    size_t bytes = sizeof(float) * n;
    float* d_in = nullptr;
    float* d_db = nullptr;
    if (int e = checkCuda(hipMalloc(&d_in, bytes), "malloc d_in")) return e;
    if (int e = checkCuda(hipMalloc(&d_db, bytes), "malloc d_db")) return e;
    if (int e = checkCuda(hipMemcpy(d_in, in_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    dim3 blk(256), grd((n + 255) / 256);
    k_power_db_real<<<grd,blk>>>(d_in, d_db, n, floor_db);
    if (int e = checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e = checkCuda(hipMemcpy(out_host, d_db, sizeof(float) * n, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d_in);
    hipFree(d_db);
    return 0;
}

int iq_fftshift_inplace(float* io_host, int n)
{
    if (!io_host || n <= 0) return 3;
    size_t bytes = sizeof(float) * 2 * n;
    float* d = nullptr;
    if (int e = checkCuda(hipMalloc(&d, bytes), "malloc d")) return e;
    if (int e = checkCuda(hipMemcpy(d, io_host, bytes, hipMemcpyHostToDevice), "H2D")) return e;

    dim3 blk(256), grd(((n / 2) + 255) / 256);
    k_fftshift_interleaved<<<grd,blk>>>(d, n);
    if (int e = checkCuda(hipDeviceSynchronize(), "sync")) return e;

    if (int e = checkCuda(hipMemcpy(io_host, d, bytes, hipMemcpyDeviceToHost), "D2H")) return e;
    hipFree(d);
    return 0;
}

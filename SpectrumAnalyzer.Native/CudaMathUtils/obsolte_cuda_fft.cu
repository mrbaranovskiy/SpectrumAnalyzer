#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>
//chat gpt generated shit...

inline const char* cufft_err(hipfftResult r) {
    switch (r) {
    case HIPFFT_SUCCESS: return "CUFFT_SUCCESS";
    case HIPFFT_INVALID_PLAN: return "CUFFT_INVALID_PLAN";
    case HIPFFT_ALLOC_FAILED: return "CUFFT_ALLOC_FAILED";
    case HIPFFT_INVALID_TYPE: return "CUFFT_INVALID_TYPE";
    case HIPFFT_INVALID_VALUE: return "CUFFT_INVALID_VALUE";
    case HIPFFT_INTERNAL_ERROR: return "CUFFT_INTERNAL_ERROR";
    case HIPFFT_EXEC_FAILED: return "CUFFT_EXEC_FAILED";
    case HIPFFT_SETUP_FAILED: return "CUFFT_SETUP_FAILED";
    case HIPFFT_INVALID_SIZE: return "CUFFT_INVALID_SIZE";
    case HIPFFT_UNALIGNED_DATA: return "CUFFT_UNALIGNED_DATA";
    default: return "CUFFT_UNKNOWN_ERROR";
    }
}

inline int checkCuda(hipError_t st, const char* where) {
    if (st != hipSuccess) {
        std::fprintf(stderr, "[CUDA] %s: %s\n", where, hipGetErrorString(st));
        return 1;
    }
    return 0;
}

// i saw this in the book...
inline int checkCufft(hipfftResult st, const char* where) {
    if (st != HIPFFT_SUCCESS) {
        std::fprintf(stderr, "[cuFFT] %s: %s\n", where, cufft_err(st));
        return 2;
    }
    return 0;
}

// kernels.cu

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>

__global__ void saxpy_kernel(float a, const float* x, const float* y, float* out, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) out[i] = a * x[i] + y[i];
}


#include <cstdio>

static inline int checkCuda(hipError_t st, const char* where){
    if (st != hipSuccess){ std::fprintf(stderr,"[CUDA]%s: %s\n",where,hipGetErrorString(st)); return 1; }
    return 0;
}
static inline int checkCufft(hipfftResult st, const char* where){
    if (st != HIPFFT_SUCCESS){ std::fprintf(stderr,"[cuFFT]%s: %d\n",where,st); return 2; }
    return 0;
}

__global__ void k_power(const float* __restrict__ x, float* __restrict__ p, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // i is complex index
    if (i >= n) return;
    float re = x[2*i+0];
    float im = x[2*i+1];
    p[i] = re*re + im*im; // power
}

__global__ void k_power_db(const float* __restrict__ x, float* __restrict__ p_db, int n, float floor_db){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    float re = x[2*i+0];
    float im = x[2*i+1];
    float pw = re*re + im*im;
    // 10*log10(power). add tiny epsilon; clamp
    float db = 10.0f * log10f(fmaxf(pw, 1e-30f));
    p_db[i] = fmaxf(db, floor_db);
}

__global__ void k_scale(float* __restrict__ freqs, int N, float Fs)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < N) {
        freqs[k] = k * (Fs / N);
    }
}

// in-place fftshift for interleaved complex buffer
__global__ void k_fftshift_interleaved(float* x, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x; // complex index
    int half = n/2;
    if (i >= half) return;
    int j = i + half + (n & 1); // odd-N handling
    if (j >= n) j -= n;         // wrap
    // swap complex pairs
    float r0 = x[2*i], im0 = x[2*i+1];
    float r1 = x[2*j], im1 = x[2*j+1];
    x[2*i] = r1; x[2*i+1] = im1;
    x[2*j] = r0; x[2*j+1] = im0;
}
